#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cstring>
#include <fstream>
#include <iostream>
#include <vector>



void SequentialCalculation(const int &n,
                           const int &m,
                           const std::vector<std::vector<int>> &A,
                           const std::vector<std::vector<int>> &B,
                           std::vector<std::vector<int>> *C) {

  std::vector<std::vector<int>> B_power, next_B_power;
  std::vector<std::vector<int>> D;
  (*C) = A;
  B_power = B;
  int tmp;
  for (int t = 1; t<=m; t++) {
    D = std::vector<std::vector<int>>(n, std::vector<int>(n,0));
    for (int i = 0; i<n; i++) {
      for (int j = 0; j<n; j++) {
        for (int k = 0; k<n; k++) {
          D[i][j] = (D[i][j] + A[i][k] * B_power[k][j])%2;
        }
      } 
    }
    for (int i = 0; i<n; i++) {
      for (int j = 0; j<n; j++) {
        (*C)[i][j] = ((*C)[i][j] + D[i][j]) %2; 
      }
    } 
    if (t==m)
      break;
    next_B_power = std::vector<std::vector<int>>(n, std::vector<int>(n,0));
    for (int i = 0; i<n; i++) {
      for (int j = 0; j<n; j++) {
        for (int k = 0; k<n; k++)
          next_B_power[i][j] = (next_B_power[i][j]+ B_power[i][k]*B[k][j])%2;
      } 
    }
    B_power = next_B_power;
  }
}

bool LoadFile(const std::string &input_file_path, int *n, int *m, std::vector<std::vector<int>> *A,
              std::vector<std::vector<int>> *B) {
  std::ifstream fin(input_file_path.c_str());
  if (!fin.is_open()) {
    return false;
  }
  fin >> (*n) >> (*m);
  *A = std::vector<std::vector<int>>(*n,std::vector<int>(*n,0));
  *B = std::vector<std::vector<int>>(*n,std::vector<int>(*n,0));
  for (int i = 0;i < (*n); i++)
    for (int j = 0;j < (*n); j++)
      fin >> (*A)[i][j];
  for (int i = 0;i < (*n); i++)
    for (int j = 0;j < (*n); j++)
      fin >> (*B)[i][j];
  fin.close();
  return true;
}

void TestAnswerCorrectness(const std::vector<std::vector<int>> &sequential_answer,
                           const std::vector<std::vector<int>> &parallel_answer) {
  if (sequential_answer.size() != parallel_answer.size()) {
    std::cout << "Error! The number of sequential_answer and parallel_answer "
                 "is not the same"
              << std::endl;
    return ;
  }
  long long sum_sequential_answer = 0;
  long long sum_parallel_answer = 0;
  int sum_error = 0;
  for (uint i = 0; i < sequential_answer.size(); i++) {
    if (sequential_answer[i].size() != parallel_answer[i].size())
    {
      std::cout << "Error! The number of sequential_answer and parallel_answer "
                 "is not the same"
              << std::endl;
      return ;
    }
    for (uint j = 0; j < sequential_answer[i].size(); j++) {
      sum_error +=  abs(sequential_answer[i][j] - parallel_answer[i][j]);
      sum_sequential_answer += sequential_answer[i][j];
      sum_parallel_answer += parallel_answer[i][j];  
    }
  }
  std::cout << "sum_sequential_answer = " << sum_sequential_answer << std::endl;
  std::cout << "sum_parallel_answer = " << sum_parallel_answer << std::endl;

  if (sum_error > 0) {
    std::cout << "Wrong Answer" << std::endl;
  } else {
    std::cout << "Correct!!!" << std::endl;
  }
}

// ==============================================================
// ====    Write your functions below this line    ====
// ==============================================================
// ==============================================================

__global__ void Kernel(int* A, int* B, int* C, int m, int thread_martix_row, int thread_martix_col, int* Iter_AB)
{
    // 获取当前线程信息
	int bx = blockIdx.x;
	int tx = threadIdx.x;

    // 块个数和线程个数
    int bn = gridDim.x;
    int tn = blockDim.x;

    // 计算每个线程计算的区域范围
    int thread_row = ceil((1.0 * thread_martix_row) / bn);
    int thread_col = ceil((1.0 * thread_martix_col) / tn);

    // 传值
    for (int i = bx * thread_row; i < (bx + 1) * thread_row; i++){
		for (int j = tx * thread_col; j < (tx + 1) * thread_col; j++){
			if (i < thread_martix_row && j < thread_martix_col){
                Iter_AB[i * thread_martix_col + j] = A[i * thread_martix_col + j];
                C[i * thread_martix_col + j] = A[i * thread_martix_col + j];
			}
        }
    }
	__syncthreads();

    // 计算
    for (int t = 0; t < m; t++){
        
        // 迭代部分 A = A = A*B = A*B*B = ......
        for (int i = bx * thread_row; i < (bx + 1) * thread_row; i++){
		    for (int j = tx * thread_col; j < (tx + 1) * thread_col; j++){
			    if (i < thread_martix_row && j < thread_martix_col){
                    A[i * thread_martix_col + j] = Iter_AB[i * thread_martix_col + j]; 
			    }
            }
        }
        __syncthreads();

        //计算A = A = A*B = A*B*B = ......和B的乘积（前序列乘B）放入迭代缓存部分
        for (int i = bx * thread_row; i < (bx + 1) * thread_row; i++){
		    for (int j = tx * thread_col; j < (tx + 1) * thread_col; j++){
			    if (i < thread_martix_row && j < thread_martix_col){
                    int sum = 0;
                    for (int k = 0; k < thread_martix_col; k++){
                        sum += A[i * thread_martix_col + k] * B[k * thread_martix_col + j];
                    }
                    Iter_AB[i * thread_martix_col + j] = sum % 2;
			    }
            }
        }
		__syncthreads();

        //C = A * (A*B) * (A*B*B) * ......
        for (int i = bx * thread_row; i < (bx + 1) * thread_row; i++){
		    for (int j = tx * thread_col; j < (tx + 1) * thread_col; j++){
			    if (i < thread_martix_row && j < thread_martix_col){
                    C[i * thread_martix_col + j] = ( C[i * thread_martix_col + j] + Iter_AB[i * thread_martix_col + j] ) % 2;
			    }
            }
        }
        __syncthreads();
    }

}

void parallelCalculation(int &n,
                         int &m,
                         const std::vector<std::vector<int>> &A,
                         const std::vector<std::vector<int>> &B,
                         std::vector<std::vector<int>> *C,
                         int number_of_processes,
                         int rank,
                         int number_of_block_in_a_grid,
                         int number_of_thread_in_a_block) {

    int new_martix_n;
    int thread_martix_row;
    int* MatrixA_h, * MatrixB_h, * MatrixC_h;
	int* MatrixB_d;
	int* LocalMatrixA_h = nullptr, * LocalMatrixC_h = nullptr;
	int* LocalMatrixA_d = nullptr, * LocalMatrixC_d = nullptr;
	int* Matrix_tmp_d = nullptr;
    if (rank == 0){
        new_martix_n = ceil((1.0 * n) / number_of_processes) * number_of_processes;
        thread_martix_row = new_martix_n / number_of_processes;
    	MatrixA_h = (int*)malloc(new_martix_n * n * sizeof(int));
		MatrixB_h = (int*)malloc(new_martix_n * n * sizeof(int));
		MatrixC_h = (int*)malloc(new_martix_n * n * sizeof(int));
        for(int i = 0; i < n; i++){
            for(int j = 0; j < n; j++){
                MatrixA_h[i * n + j] = A[i][j];
                MatrixB_h[i * n + j] = B[i][j];
            }
        }
    }
    MPI_Barrier(MPI_COMM_WORLD);
	
    //广播
	MPI_Bcast(&new_martix_n, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&m, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&n, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&thread_martix_row, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&number_of_block_in_a_grid, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&number_of_thread_in_a_block, 1, MPI_INT, 0, MPI_COMM_WORLD);
    
  LocalMatrixA_h = (int*)malloc(thread_martix_row * n * sizeof(int));
	LocalMatrixC_h = (int*)malloc(thread_martix_row * n * sizeof(int));

	MPI_Scatter(MatrixA_h, thread_martix_row * n, MPI_INT, LocalMatrixA_h, thread_martix_row * n, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Barrier(MPI_COMM_WORLD);

	if (rank != 0) {
        MatrixB_h = (int*)malloc(new_martix_n * n * sizeof(int));
    }
	MPI_Bcast(MatrixB_h, new_martix_n * n, MPI_INT, 0, MPI_COMM_WORLD);

	//把矩阵放到gpu上
	hipMalloc(&MatrixB_d, new_martix_n * n * sizeof(int));
	hipMalloc(&LocalMatrixA_d, thread_martix_row * n * sizeof(int));
	hipMalloc(&Matrix_tmp_d, thread_martix_row * n * sizeof(int));
	hipMalloc(&LocalMatrixC_d, thread_martix_row * n * sizeof(int));
	hipMemcpy(MatrixB_d, MatrixB_h, new_martix_n * n * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(LocalMatrixA_d, LocalMatrixA_h, thread_martix_row * n * sizeof(int), hipMemcpyHostToDevice);

	//cuda部分
	dim3 grid(number_of_block_in_a_grid);
	dim3 block(number_of_thread_in_a_block);
	Kernel << <grid, block>> > (LocalMatrixA_d, MatrixB_d, LocalMatrixC_d, m, thread_martix_row, n, Matrix_tmp_d);
	hipMemcpy(LocalMatrixC_h, LocalMatrixC_d, thread_martix_row * n * sizeof(int), hipMemcpyDeviceToHost);
	MPI_Gather(LocalMatrixC_h, thread_martix_row * n, MPI_INT, MatrixC_h, thread_martix_row * n, MPI_INT, 0, MPI_COMM_WORLD);

	//mem转到vec
    if (rank == 0){
        *C = std::vector<std::vector<int>>(n, std::vector<int>(n, 0));
        for (int i = 0; i < n; i++){
            for (int j = 0; j < n; j++){
                (*C)[i][j] = MatrixC_h[i * n + j];
            }
        }
    }
}

// ==============================================================
// ====    Write your functions above this line    ====
// ==============================================================
// ==============================================================


int main(int argc, char **argv) {
  int number_of_processes, rank;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &number_of_processes);
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);
  double parallel_start_time;

  int number_of_block_in_a_grid;
  int number_of_thread_in_a_block;
  int n,m;
  std::vector<std::vector<int>> A;
  std::vector<std::vector<int>> B;
  if (rank == 0) {
    if (argc < 4) {
      std::cout << "Error! Please use \"mpiexec -n [process number] "
                   "[--hostfile hostfile] multiple [number_of_block_in_a_grid] [number_of_thread_in_a_block] [data_file_name]\"\n";
      return 1;
    } else {
      number_of_block_in_a_grid = std::atoi(argv[1]);
      number_of_thread_in_a_block = std::atoi(argv[2]);
      std::string input_file_path = std::string(argv[3]);
      std::cout << "number_of_block_in_a_grid:" << number_of_block_in_a_grid<< std::endl;
      std::cout << "number_of_thread_in_a_block:" << number_of_thread_in_a_block<< std::endl;
      if (!LoadFile(input_file_path, &n, &m, &A, &B)) {
        std::cout << "Error! Please check the format of input file\n";
        return 1;
      }
    }
  }
  std::vector<std::vector<int>> parallel_answer;

  if (rank == 0) {
    parallel_start_time = MPI_Wtime();
  }
  
  // ==============================================================
  // ====    Write your implementation below this line    ====
  // ==============================================================
  // ==============================================================

  parallelCalculation(n, m, A, B, &parallel_answer, number_of_processes, rank, number_of_block_in_a_grid, number_of_thread_in_a_block);



  // ==============================================================
  // ====    Write your implementation above this line    ====
  // ==============================================================
  // ==============================================================
  MPI_Barrier(MPI_COMM_WORLD);
  if (rank == 0) {
    double parallel_end_time = MPI_Wtime();
    double parallel_running_time = parallel_end_time - parallel_start_time;
    std::cout << "parallel running time:" << parallel_running_time << std::endl;
    std::vector<std::vector<int>> sequential_answer;
    double sequential_start_time = MPI_Wtime();

    SequentialCalculation(n, m, A, B, &sequential_answer);
    double sequential_end_time = MPI_Wtime();
    double sequential_running_time =
        sequential_end_time - sequential_start_time;
    std::cout << "sequential running time:" << sequential_running_time
              << std::endl;
    std::cout << "speed up:" <<  sequential_running_time/parallel_running_time
              << std::endl;
    TestAnswerCorrectness(sequential_answer, parallel_answer);
  }
  MPI_Finalize();
  return 0;
}