#include "hip/hip_runtime.h"
#include <mpi.h> 
#include <stdio.h> 
#include <stdlib.h> 
#include <unistd.h>
#include <time.h>
#include <omp.h>
#include <iostream>

__global__ void VectorAddition(int *a_device_this_process, int *b_device_this_process, int *c_device_this_process, int elements_per_process)
{  	
	int block_num = gridDim.x;
  	int block_id = blockIdx.x;
  	int thread_num = blockDim.x;
  	int thread_id = threadIdx.x;
    
	int total_thread_num = block_num*thread_num;
	int id = block_id * thread_num + thread_id;
    for (int i = id; i <elements_per_process; i+=total_thread_num)
	{
		c_device_this_process[i]= a_device_this_process[i] + b_device_this_process[i];
	}
}

int main(int argc, char* argv[]) 
{

	int provided, pid, np;			 

	int *a, *b, *c;
	
	srand(time(NULL));

	MPI_Init_thread(&argc, &argv, MPI_THREAD_FUNNELED, &provided);

	MPI_Comm_rank(MPI_COMM_WORLD, &pid);
	MPI_Comm_size(MPI_COMM_WORLD, &np);

	int total_num, elements_per_process, n_elements_recieved;
	int block_num, thread_num;
    if (pid == 0) {
		block_num = 3; 
		thread_num = 2;
		elements_per_process = block_num*thread_num * (rand() % 5 + 1);
		total_num = np * elements_per_process;
				
		a = (int *)malloc(total_num * sizeof(int));
        b = (int *)malloc(total_num * sizeof(int));
        c = (int *)malloc(total_num * sizeof(int));
        for (int i = 0; i < total_num; i++) {
			a[i] = rand() % 10;
            b[i] = rand() % 10;
        }
	}

	int *a_device, *b_device, *c_device;
	int *a_device_this_process, *b_device_this_process, *c_device_this_process;
	if (pid==0) {
		hipMalloc( (void **)&a_device, total_num * sizeof(int));
		hipMalloc( (void **)&b_device, total_num*sizeof(int));
		hipMalloc( (void **)&c_device, total_num * sizeof(int));
		hipMemcpy( (void *)a_device, (void *)a, total_num * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy( (void *)b_device, (void *)b, total_num * sizeof(int), hipMemcpyHostToDevice);
	}

	MPI_Bcast(&elements_per_process, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&block_num, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&thread_num, 1, MPI_INT, 0, MPI_COMM_WORLD);

	hipMalloc( (void **)&a_device_this_process, elements_per_process * sizeof(int));
	hipMalloc( (void **)&b_device_this_process, elements_per_process*sizeof(int));
	hipMalloc( (void **)&c_device_this_process, elements_per_process * sizeof(int));
	
	MPI_Scatter(a_device, elements_per_process, MPI_INT, 
				a_device_this_process, elements_per_process, MPI_INT,
				0, MPI_COMM_WORLD); 
    MPI_Scatter(b_device, elements_per_process, MPI_INT, 
				b_device_this_process, elements_per_process, MPI_INT,
				0, MPI_COMM_WORLD); 

	VectorAddition<<<block_num, thread_num>>>(a_device_this_process, b_device_this_process, c_device_this_process, elements_per_process);	

	MPI_Gather(c_device_this_process, elements_per_process, MPI_INT,
               c_device, elements_per_process, MPI_INT, 0,
               MPI_COMM_WORLD); 

	if (pid == 0) {
		hipMemcpy( (void *)c, (void *)c_device, total_num * sizeof(int), hipMemcpyDeviceToHost);	
	}

	if (pid==0) {
		hipFree(a_device);
		hipFree(b_device);
		hipFree(c_device);
	}
	hipFree(a_device_this_process);
	hipFree(b_device_this_process);
	hipFree(c_device_this_process);

	if (pid == 0) {
		printf("A:");
		for (int i = 0; i < total_num ; i++) {
			printf("%2d ",a[i]);
		}
		printf("\n");

		printf("B:");
		for (int i = 0; i < total_num ; i++) {
			printf("%2d ",b[i]);
		}
		printf("\n");

		printf("C:");
		for (int i = 0; i < total_num ; i++) {
			printf("%2d ",c[i]);
		}
		printf("\n");

		free(a);
		free(b);
		free(c);
	}


	MPI_Finalize(); 

	return 0; 
} 
